#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

/*for Sort-free
// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}
*/
__global__ void duplicateToTile(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
  uint32_t* counters,
	uint32_t* gaussian_idx,
  uint32_t* tile_idx,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;


	if (radii[idx] > 0)
	{
		uint2 rect_min, rect_max;
		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
        uint32_t tile_idx = y * grid.x + x;
        uint32_t off = counters[tile_idx];
        uint32_t glb_inst_pos = (tile_idx == 0) ? 0 : offsets[tile_idx - 1];
        gaussian_idx[glb_inst_pos + off] = idx;
        tile_idx[glb_inst_pos + off] = tile_idx;

        atomicAdd(&counters[tile_idx], 1);
			}
		}
	}
}

/* for Sort-free
// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}
*/
__global__ void identifyTileRanges(int L, uint32_t* point_list_tile, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint32_t currtile = point_list_tile[idx];
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_tile[idx - 1];
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
  }



// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
  /* for Sort-free
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
  */
  obtain(chunk, geom.depthweights, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
  obtain(chunk, binning.point_list_idx, P, 128);
  obtain(chunk, binning.point_list_tile, P, 128);
	return binning;
}

// for Sort-free
CudaRasterizer::TileState CudaRasterizer::TileState::fromChunk(char*& chunk, size_t N)
{
	TileState tile;
  obtain(chunk, tile.tile_point_touched, N, 128);
  obtain(chunk, tile.tile_offsets, N, 128);
  obtain(chunk, tile.tile_counters, N, 128);
  hipcub::DeviceScan::InclusiveSum(nullptr, tile.tile_scan_size, tile.tile_point_touched, tile.tile_point_touched, N);
  obtain(chunk, tile.tile_scanning_space, N, 128);

  CHECK_CUDA(hipMemset(tile.tile_counters, 0, N * sizeof(uint32_t)), debug);

  return tile;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
  std::function<char* (size_t)> tileBuffer, // for Sort-free
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
  const float background_weight,// for Sort-free
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	//const float* opacities,
  const float* shs_opacities, //for Sort-free
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float sigma, // for Sort-free
	const float* v_depthweight, // for Sort-free
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	int* radii,
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);


	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

  const int Tiles_Num = tile_grid.x * tile_grid.y;

  // for Sort-free
  size_t tile_chunk_size = required<TileState>(Tiles_Num);
  char* tile_chunkptr = tileBuffer(tile_chunk_size);
  TileState tileState = TileState::fromChunk(tile_chunkptr, Tiles_Num);


	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
    sigma,
    v_depthweight,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
    geomState.depthweights,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		tileState.tile_point_touched,
		prefiltered
	), debug)

  /*for Sort-free
	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);
  */
  CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(tileState.tile_scanning_space,tileState.tile_scan_size,tileState.tile_point_touched,tileState.tile_offsets,Tiles_Num), debug)

  int num_rendered;

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

  /* for Sort-free
	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)
  */

  duplicateToTile << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		tileState.tile_offsets,
    tileState.tile_counters,
		binningState.point_list_idx
    binningState.point_list_tile,
    radii,
		tile_grid)
	CHECK_CUDA(, debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

  /* for Sort-free
	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_idx_key,
			imgState.ranges);
	CHECK_CUDA(, debug)
  */
  if (num_rendered > 0)
    identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
      num_rendered,
      binningState.point_list_tile,
      imgState.ranges);
  CHECK_CUDA(, debug)


	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list_idx,
		width, height,
		geomState.means2D,
		feature_ptr,
    geomState.depthweights,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
    background_weight,
		out_color), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor), debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot), debug)
}